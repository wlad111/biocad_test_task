#include "hip/hip_runtime.h"
#include "sum.cuh"
#include <cstdio>
#include <iostream>

const float COEFFICIENT = 1389.38757;

int get_max_cols(Matrix A) {
    int globalsum = 0;
    int n = A.height;

    for (size_t i = 0; i < n; i++) {
        int localsum = 0;
        for (size_t j = 0; j < n; j++) {
            if (A.elements[i * n + j] > 0) {
                localsum++;
            }
        }
        if (localsum > globalsum) {
            globalsum = localsum;
        }
    }
    return globalsum;
}

void fill_ell_matrix(ELL_Matrix ell, Matrix A, int nrows, int ncols, int ncols_per_row) {



    for (size_t i = 0; i < nrows; i++) {
        int colidx = 0;
        for (size_t j = 0; j < ncols; j++) {
            if (A.elements[i * nrows + j] != 0) {
                ell.data[colidx * nrows + i] = A.elements[i * nrows + j];
                ell.col_indices[colidx * nrows + i] = j;
                colidx++;
            }
        }
    }
}

void ell_mallocHost(){

}

void ell_freeHost() {

}

__device__ float sign(float x) {
    float t = x < 0 ? -1.0 : 0.0;
    return x > 0 ? 1.0 : t;
}

__device__ float atom_dist (atom a1, atom a2) {
    float sum =  (a1.x - a2.x) * (a1.x - a2.x) +
            (a1.y - a2.y) * (a1.y - a2.y) +
            (a1.z - a2.z) * (a1.z - a2.z);
    return sqrtf(sum);
}

void calculate(float* bonds, size_t n, atom* atom_coords, float* charges) {
    //TODO hard-coded value! fix it
    size_t ncols_per_row = 4;

    Matrix A;
    A.elements = bonds;
    A.height = n;
    A.width = n;

    size_t size = A.width * A.height * sizeof(float);

    ELL_Matrix ell_A;

    size_t sizefloat = n * ncols_per_row * sizeof(float);
    size_t sizeint = n * ncols_per_row * sizeof(int);

    hipHostMalloc((void**)&ell_A.data, sizefloat);
    hipHostMalloc((void**)&ell_A.col_indices, sizeint);

    ell_A.ncols = n;
    ell_A.nrows = n;
    ell_A.ncols_per_row = ncols_per_row;

    fill_ell_matrix(ell_A, A, n, n, 4);

    ELL_Matrix d_ell_A;
    d_ell_A.ncols_per_row = ell_A.ncols_per_row;
    d_ell_A.ncols = ell_A.ncols;
    d_ell_A.nrows = ell_A.nrows;

    hipMalloc((void**)&d_ell_A.data, sizefloat);
    hipMalloc((void**)&d_ell_A.col_indices, sizeint);

    hipMemcpy(d_ell_A.data, ell_A.data, sizefloat, hipMemcpyHostToDevice);
    hipMemcpy(d_ell_A.col_indices, ell_A.col_indices, sizeint, hipMemcpyHostToDevice);

    Matrix Asquared;
    Asquared.width = n;
    Asquared.height = n;
    hipHostMalloc((void**)&Asquared.elements, size);

    /*Matrix Acubed;
    hipHostMalloc((void**)&Acubed.elements, size);
    Acubed.height = n;
    Acubed.width = n;*/

    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = A.width;
    d_B.height = A.height;
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_Asquared;
    d_Asquared.width = A.width;
    d_Asquared.height = A.height;
    hipMalloc((void**)&d_Asquared.elements, size);



    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    dim3 dimGrid(A.width / dimBlock.x + 1, A.height / dimBlock.y + 1);

    dim3 dimBlock1(BLOCK_SIZE);

    dim3 dimGrid1(n / dimBlock1.x + 1);
    //std::cout << "Launching kernel for multiply" << std::endl;

    //matrix_mult<<<dimGrid, dimBlock>>>(d_A, d_B, d_Asquared);
    //hipDeviceSynchronize();
    //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;


    for (int i = 0; i < n; i++) {
        ell_mult<<<dimGrid1, dimBlock1>>>(d_ell_A, d_A, d_Asquared, i);
        //hipDeviceSynchronize();
        //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    }
    hipDeviceSynchronize();
    hipMemcpy(Asquared.elements, d_Asquared.elements, size, hipMemcpyDeviceToHost);

    int ncols_per_row2 = get_max_cols(Asquared);

    ELL_Matrix ell_Asq;

    size_t sizefloat2 = n * ncols_per_row2 * sizeof(float);
    size_t sizeint2 = n * ncols_per_row2 * sizeof(int);

    hipHostMalloc((void**)&ell_Asq.data, sizefloat2);
    hipHostMalloc((void**)&ell_Asq.col_indices, sizeint2);

    ell_Asq.ncols = n;
    ell_Asq.nrows = n;
    ell_Asq.ncols_per_row = ncols_per_row2;

    fill_ell_matrix(ell_Asq, Asquared, n, n, ncols_per_row2);

    ELL_Matrix d_ell_Asq;
    d_ell_Asq.ncols_per_row = ell_Asq.ncols_per_row;
    d_ell_Asq.ncols = ell_Asq.ncols;
    d_ell_Asq.nrows = ell_Asq.nrows;

    hipMalloc((void**)&d_ell_Asq.data, sizefloat2);
    hipMalloc((void**)&d_ell_Asq.col_indices, sizeint2);

    hipMemcpy(d_ell_Asq.data, ell_Asq.data, sizefloat2, hipMemcpyHostToDevice);
    hipMemcpy(d_ell_Asq.col_indices, ell_Asq.col_indices, sizeint2, hipMemcpyHostToDevice);

    Matrix d_Acubed;
    d_Acubed.width = A.width;
    d_Acubed.height = A.height;
    hipMalloc((void**)&d_Acubed.elements, size);

    atom* d_atoms;
    hipMalloc((void**)&d_atoms, n * sizeof(atom));
    hipMemcpy(d_atoms, atom_coords, n * sizeof(atom), hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++) {
        ell_mult<<<dimGrid1, dimBlock1>>>(d_ell_Asq, d_A, d_Acubed, i);
        //hipDeviceSynchronize();
        //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    //matrix_mult<<<dimGrid, dimBlock>>>(d_A, d_Asquared, d_Acubed);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    hipDeviceSynchronize();
    std::cout << "Finished multiplication" << std::endl;
    //hipMemcpy(Acubed.elements, d_Acubed.elements, size, hipMemcpyDeviceToHost);

    Matrix F;
    F.width = n;
    F.height = n;

    hipMalloc((void**)&F.elements, size);

    get_matrix_F<<<dimGrid, dimBlock>>>(d_Acubed, d_Asquared, d_A, d_atoms, F);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

/*    Matrix F_host;
    F_host.width = F.width;
    F_host.height = F.width;
    hipHostMalloc((void**)&F_host.elements, size);
    hipMemcpy(F_host.elements, F.elements, size, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < n * n; i++) {
        if (F_host.elements[i] != 0) {
            std::cout << F_host.elements[i] << std::endl;
        }
    }*/

    float* res;
    float* d_res;
    float* d_charges;



    hipHostMalloc((void**)&res, n * sizeof(float));
    hipMalloc((void**)&d_res, n * sizeof(float));

    hipMalloc((void**)&d_charges, n * sizeof(float));
    hipMemcpy(d_charges, charges, n * sizeof(float), hipMemcpyHostToDevice);


  /*  dim3 dimBlock1(BLOCK_SIZE);

    dim3 dimGrid1(n / dimBlock1.x + 1);*/

    get_vector_to_sum<<<dimGrid1, dimBlock1>>>(F, d_charges, d_res);
    hipDeviceSynchronize();
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;



    hipMemcpy(res, d_res, n * sizeof(float), hipMemcpyDeviceToHost);

    float ans = 0;


    for (size_t i = 0; i < n; i++) {
        ans += res[i];
    }

    std::cout << "Answer is " << (COEFFICIENT * ans) / 2 << std::endl;

    hipFree(d_A.elements);
    hipFree(d_Asquared.elements);
    hipFree(d_Acubed.elements);
    hipFree(d_atoms);
    hipFree(d_res);
    hipFree(d_charges);
}




__global__ void matrix_mult(Matrix A, Matrix B, Matrix C){
    float Cvalue = 0;
    if (blockIdx.y * blockDim.y + threadIdx.y < A.width && blockIdx.x * blockDim.x + threadIdx.x < A.width){
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        for (int e = 0; e < A.width; ++e) {
            Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
        }
    C.elements[row * C.width + col] = Cvalue;
    }
}

__global__ void get_matrix_F(Matrix ACube, Matrix ASquare, Matrix A, atom* atoms, Matrix F) {
    float value = 0;
    float cube = 0;
    float square = 0;
    float a = 0;
    if (blockIdx.y * blockDim.y + threadIdx.y < A.width && blockIdx.x * blockDim.x + threadIdx.x < A.width){
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int idx = row * A.width + col;
        if (row != col) {
            value = 1 - 0.5 * sign(ACube.elements[idx] + ASquare.elements[idx] + A.elements[idx]) -
                              0.5 * sign(ASquare.elements[idx] + A.elements[idx]);
            value /= atom_dist(atoms[row], atoms[col]);
            F.elements[idx] = value;
        }
        else {
            F.elements[idx] = 0.0;
        }
    }
}

__global__ void get_vector_to_sum(Matrix F, float* q, float* res) {
    float value = 0;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < F.width) {
        for (int j = 0; j < F.height; j++) { //TODO maybe permute indices order
            value += F.elements[j * F.height + i] * q[j];
        }
        value *= q[i];
        res[i] = value;
    }
}

/**
 * kernel for ELL matrix-vector multiplication
 *
 */

__global__ void ell_mult(ELL_Matrix A, Matrix B, Matrix C, int i){
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < A.nrows) {
        float dot = 0;

        for (int idx = 0; idx < A.ncols_per_row; idx++) {
            int col = A.col_indices[A.nrows * idx + row];
            float val = A.data[A.nrows * idx + row];

            if (val != 0) {
                dot += val * B.elements[i * B.height + col];
            }
            C.elements[i * C.height + row] = dot;
        }
    }
}





